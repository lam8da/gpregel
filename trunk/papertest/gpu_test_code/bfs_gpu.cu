#include "hip/hip_runtime.h"
#include <algorithm>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <fstream>
#include <iostream>
#include <map>
#include <set>
#include <string>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#include "bfs.h"

using std::cerr;
using std::cin;
using std::cout;
using std::endl;
using std::ifstream;
using std::map;
using std::ofstream;
using std::set;
using std::string;
using std::vector;

__global__ void BfsKernel(
    const unsigned int ss,
    const unsigned int numv,
    const unsigned int *va,
    const unsigned int *ea,
    bool *fa,
    bool *xa,
    unsigned int *ca) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < numv) {
    if (fa[tid]) {
      fa[tid] = false;
      // xa[tid] = true;
      const unsigned int end = va[tid];
      for (int i = (tid == 0 ? 0 : va[tid - 1]); i < end; ++i) {
        const unsigned int nid = ea[i];
        if (ca[nid] > ss + 1) {
          ca[nid] = ss + 1;
          // fa[nid] = true;  // Conflict with line 41!
          xa[nid] = true;
          fa[numv] = true;  // tell host that the iteration has not finished
        }
      }
    }
  }
}

__global__ void ResetFaXaKernel(const unsigned int numv, bool *fa, bool *xa) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < numv) {
    if (xa[tid]) {
      fa[tid] = true;
      xa[tid] = false;
    }
  }
}

unsigned int Bfs(
    const unsigned int numv,
    const unsigned int nume,
    const unsigned int root,
    const unsigned int *h_va,
    const unsigned int *h_ea,
    unsigned int *h_ca,
    float *duration) {
  unsigned int *d_va = NULL;
  unsigned int *d_ea = NULL;
  unsigned int *d_ca = NULL;
  bool *d_fa = NULL;
  bool *d_xa = NULL;

  const unsigned int vsize = numv * sizeof(unsigned int);
  const unsigned int esize = nume * sizeof(unsigned int);
  const unsigned int bsize = numv * sizeof(bool);

  checkCudaErrors(hipMalloc((void **)&d_va, vsize));
  checkCudaErrors(hipMalloc((void **)&d_ea, esize));
  checkCudaErrors(hipMalloc((void **)&d_ca, vsize));
  checkCudaErrors(hipMalloc((void **)&d_fa, bsize + sizeof(bool)));
  checkCudaErrors(hipMalloc((void **)&d_xa, bsize));

  checkCudaErrors(hipMemcpy(d_va, h_va, vsize, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_ea, h_ea, esize, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemset(d_ca, 0xff, vsize));
  checkCudaErrors(hipMemset(d_fa, 0, bsize + sizeof(bool)));
  checkCudaErrors(hipMemset(d_xa, 0, bsize));

  bool boolean = true;
  checkCudaErrors(hipMemcpy(d_fa + root, &boolean, sizeof(bool), hipMemcpyHostToDevice));
  unsigned int rootlvl = 0;
  checkCudaErrors(hipMemcpy(d_ca + root, &rootlvl, sizeof(unsigned int), hipMemcpyHostToDevice));

  const unsigned int threads_per_block = 128;
  const unsigned int blocks_per_grid =(numv + threads_per_block - 1) / threads_per_block;

  // ------------------------- start profiling -------------------------
  StopWatchInterface *timer;
  sdkCreateTimer(&timer);
  sdkResetTimer(&timer);
  sdkStartTimer(&timer);

  unsigned int superstep = 0;
  while (true) {
    boolean = false;
    checkCudaErrors(hipMemcpy(d_fa + numv, &boolean, sizeof(bool), hipMemcpyHostToDevice));

    BfsKernel<<<blocks_per_grid, threads_per_block>>>(superstep, numv, d_va, d_ea, d_fa, d_xa, d_ca);
    ResetFaXaKernel<<<blocks_per_grid, threads_per_block>>>(numv, d_fa, d_xa);
    ++superstep;

    checkCudaErrors(hipMemcpy(&boolean, d_fa + numv, sizeof(bool), hipMemcpyDeviceToHost));
    if (!boolean) break;
  }

  sdkStopTimer(&timer);
  *duration = sdkGetTimerValue(&timer);
  sdkDeleteTimer(&timer);
  // ------------------------- end profiling -------------------------

  checkCudaErrors(hipMemcpy(h_ca, d_ca, vsize, hipMemcpyDeviceToHost));

  checkCudaErrors(hipFree(d_va));
  checkCudaErrors(hipFree(d_ea));
  checkCudaErrors(hipFree(d_ca));
  checkCudaErrors(hipFree(d_fa));
  checkCudaErrors(hipFree(d_xa));
  return superstep;
}

//argv[1]: gpu_id
int main(int argc, char **argv) {
  unsigned int gpuid = atoi(argv[1]);
  checkCudaErrors(hipSetDevice(gpuid));

  HostGraphGlobal global;
  vector<HostGraphVertex> vertex_vec;
  vector<HostGraphEdge> edge_vec;

  Read(global, vertex_vec, edge_vec);

  // ------------------------- start cpu -------------------------
  // std::sort(vertex_vec.begin(), vertex_vec.end());
  std::sort(edge_vec.begin(), edge_vec.end());
  unsigned int max_level = CpuAlgorithm(
      999999999, global, vertex_vec, edge_vec);

  // ------------------------- start gpu -------------------------
  unsigned int *h_va = new unsigned int[global.num_vertex];
  unsigned int *h_ea = new unsigned int[global.num_edge];
  unsigned int *h_ca = new unsigned int[global.num_vertex];
  for (unsigned int i = 0; i < global.num_vertex; ++i) {
    h_va[i] = vertex_vec[i].sum_out_edge_count;
  }
  for (unsigned int i = 0; i < global.num_edge; ++i) {
    h_ea[i] = edge_vec[i].to;
  }

  float duration = 0;
  unsigned int superstep = Bfs(global.num_vertex, global.num_edge, global.root, h_va, h_ea, h_ca, &duration);

  // ------------------------- check result -------------------------
  bool correct = true;
  for (unsigned int i = 0; i < global.num_vertex; ++i) {
    if (h_ca[i] != vertex_vec[i].level) {
      correct = false;

      ofstream out("gpuout-456767321776");
      out << "gpu result:" << endl;
      for (unsigned int j = 0; j <global.num_vertex; ++j) {
        out << h_ca[j] << endl;
      }
      out << endl;
      out.close();

      out.open("cpuout-456767321776");
      out << "cpu result:" << endl;
      for (unsigned int j = 0; j <global.num_vertex; ++j) {
        out << vertex_vec[j].level << endl;
      }
      out << endl;
      out.close();
      break;
    }
  }

  delete[] h_va;
  delete[] h_ea;
  delete[] h_ca;

  // ------------------------- output -------------------------
  cout << "check result: " << (correct ? "correct, " : "WRONG!!!!! ")
       << "root: " << global.root << ", "
       << "max_level: " << max_level << ", "
       << "superstep: " << superstep << ", "
       << "gpu_duration: " << duration << " ms."
       << endl;

  checkCudaErrors(hipDeviceReset());
  return 0;
}

