// *****************************************************************************
// Filename:    message_content_manager.cc
// Date:        2013-01-08 10:06
// Author:      Guangda Lai
// Email:       lambda2fei@gmail.com
// Description: TODO(laigd): Put the file description here.
// *****************************************************************************

#include "message_content_manager.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <thrust/device_ptr.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/gather.h>

#include "constants.h"
#include "device_graph_data_types.h"
#include "device_util.h"

#ifdef LAMBDA_DEBUG
#include "debug.h"
#define LAMBDA_HEADER "------> "
#endif

#define COPY_FROM_DEVICE_TO_DEVICE( \
    FROM, TO, MEMBER, FROM_OFFSET, TO_OFFSET, COUNT, TYPE) \
    checkCudaErrors(hipMemcpyAsync( \
            TO->MEMBER + TO_OFFSET, \
            FROM.MEMBER + FROM_OFFSET, \
            COUNT * sizeof(TYPE), \
            hipMemcpyDeviceToDevice))

namespace {

unsigned int RoundUpToMultiples(
    const size_t type_size,
    const unsigned int count,
    const size_t bench_mark) {
  return (type_size * count + bench_mark - 1) / bench_mark;
}

}  // namespace

void MessageContentManager::Allocate(
    const unsigned int size,
    MessageContent *mcon) {
  mcon->d_size = size;

#ifdef LAMBDA_SHARE_ONE_MESSAGE_ARRAY

  mcon->d_space_size = 0
#ifndef LAMBDA_FULL_MESSAGE_IN_EACH_SUPERSTEP
     + RoundUpToMultiples(sizeof(bool), size, sizeof(unsigned int))
#endif

#ifdef LAMBDA_TEST_SHORTEST_PATH
     //// TODO(laigd): add user defined members
     + RoundUpToMultiples(sizeof(unsigned int), size, sizeof(unsigned int))
#else
$$M[[+ RoundUpToMultiples(sizeof(<GP_TYPE>), size, sizeof(unsigned int))]]
#endif
     ;

  ALLOCATE_ON_DEVICE(unsigned int, mcon->d_space, mcon->d_space_size);

  unsigned int offset = 0;

#ifndef LAMBDA_FULL_MESSAGE_IN_EACH_SUPERSTEP
  mcon->d_is_full = (bool*)(mcon->d_space + offset);
  offset += RoundUpToMultiples(sizeof(bool), size, sizeof(unsigned int));
#endif

#ifdef LAMBDA_TEST_SHORTEST_PATH
  //// TODO(laigd): add user defined members
  mcon->d_dist = (unsigned int*)(mcon->d_space + offset);
  offset += RoundUpToMultiples(sizeof(unsigned int), size, sizeof(unsigned int));
#else
$$M[[mcon->d_<GP_NAME> = (<GP_TYPE>*)(mcon->d_space + offset); offset += RoundUpToMultiples(sizeof(<GP_TYPE>), size, sizeof(unsigned int));]]
#endif

#else  // Not share one array

#ifndef LAMBDA_FULL_MESSAGE_IN_EACH_SUPERSTEP
  ALLOCATE_ON_DEVICE(bool,         mcon->d_is_full, mcon->d_size);
#endif

#ifdef LAMBDA_TEST_SHORTEST_PATH
  //// TODO(laigd): add user defined members
  ALLOCATE_ON_DEVICE(unsigned int, mcon->d_dist,    mcon->d_size);
#else
$$M[[ALLOCATE_ON_DEVICE(<GP_TYPE>, mcon->d_<GP_NAME>, mcon->d_size);]]
#endif

#endif  // LAMBDA_SHARE_ONE_MESSAGE_ARRAY
}

void MessageContentManager::Deallocate(MessageContent *mcon) {
#ifdef LAMBDA_SHARE_ONE_MESSAGE_ARRAY

  DEALLOCATE_ON_DEVICE(mcon->d_space);

#else  // Not share one array

#ifndef LAMBDA_FULL_MESSAGE_IN_EACH_SUPERSTEP
  DEALLOCATE_ON_DEVICE(mcon->d_is_full);
#endif

#ifdef LAMBDA_TEST_SHORTEST_PATH
  //// TODO(laigd): add user defined members
  DEALLOCATE_ON_DEVICE(mcon->d_dist);
#else
$$M[[DEALLOCATE_ON_DEVICE(mcon->d_<GP_NAME>);]]
#endif

#endif
}

void MessageContentManager::Shuffle(
    MessageContent *mcon,
    thrust::device_ptr<unsigned int> thr_shuffle_index,
    void *d_tmp_buf) {
#ifndef LAMBDA_FULL_MESSAGE_IN_EACH_SUPERSTEP
  SHUFFLE_MEMBER(bool,         mcon->d_is_full, mcon->d_size, d_tmp_buf, thr_shuffle_index);
#endif

#ifdef LAMBDA_TEST_SHORTEST_PATH
  //// TODO(laigd): add user defined members
  SHUFFLE_MEMBER(unsigned int, mcon->d_dist,    mcon->d_size, d_tmp_buf, thr_shuffle_index);
#else
$$M[[SHUFFLE_MEMBER(<GP_TYPE>, mcon->d_<GP_NAME>, mcon->d_size, d_tmp_buf, thr_shuffle_index);]]
#endif
}

void MessageContentManager::Copy(
    const MessageContent &from,
    MessageContent *to) {
#ifdef LAMBDA_SHARE_ONE_MESSAGE_ARRAY

  COPY_FROM_DEVICE_TO_DEVICE(from, to, d_space, 0, 0, from.d_space_size, unsigned int);

#else  // Not share one array

#ifndef LAMBDA_FULL_MESSAGE_IN_EACH_SUPERSTEP
  COPY_FROM_DEVICE_TO_DEVICE(from, to, d_is_full, 0, 0, from.d_size, bool        );
#endif

#ifdef LAMBDA_TEST_SHORTEST_PATH
  //// TODO(laigd): add user defined members
  COPY_FROM_DEVICE_TO_DEVICE(from, to, d_dist,    0, 0, from.d_size, unsigned int);
#else
$$M[[COPY_FROM_DEVICE_TO_DEVICE(from, to, d_<GP_NAME>, 0, 0, from.d_size, <GP_TYPE>);]]
#endif

#endif
}

#ifndef LAMBDA_FULL_MESSAGE_IN_EACH_SUPERSTEP
void MessageContentManager::Clear(MessageContent *mcon) {
  thrust::device_ptr<bool> thr_is_full(mcon->d_is_full);
  thrust::fill(thr_is_full, thr_is_full + mcon->d_size, false);
}
#endif

#ifdef LAMBDA_DEBUG
void MessageContentManager::DebugOutput(
    const MessageContent &mcon, const bool is_send_buf) {
  unsigned int *buf = NULL;
  checkCudaErrors(hipHostMalloc(&buf, mcon.d_size * sizeof(unsigned int)));

  cout << LAMBDA_HEADER << "[MessageContent "
       << (is_send_buf ? "Send" : "Recv") << "]" << endl;
#ifndef LAMBDA_FULL_MESSAGE_IN_EACH_SUPERSTEP
  DEBUG_OUTPUT(buf, mcon.d_is_full, "is_full: ", mcon.d_size, bool);
#endif

#ifdef LAMBDA_TEST_SHORTEST_PATH
  //// TODO(laigd): add user defined members
  DEBUG_OUTPUT(buf, mcon.d_dist,    "dist:    ", mcon.d_size, unsigned int);
#else
$$M[[DEBUG_OUTPUT(buf, mcon.d_<GP_NAME>, "<GP_NAME>: ", mcon.d_size, <GP_TYPE>);]]
#endif

  checkCudaErrors(hipHostFree(buf));
}
#endif

#ifdef LAMBDA_DEBUG
#undef LAMBDA_HEADER
#endif

#undef COPY_FROM_DEVICE_TO_DEVICE
