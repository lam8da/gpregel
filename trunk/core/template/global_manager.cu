// *****************************************************************************
// Filename:    global_manager.cc
// Date:        2013-01-08 10:01
// Author:      Guangda Lai
// Email:       lambda2fei@gmail.com
// Description: TODO(laigd): Put the file description here.
// *****************************************************************************

#include "global_manager.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#include "device_graph_data_types.h"
#include "generated_io_data_types.h"

#ifdef LAMBDA_DEBUG
#include "debug.h"
#define LAMBDA_HEADER "------> "
#endif

void GlobalManager::Set(const IoGlobal &src, Global *dst) {
  dst->d_num_vertex = src.num_vertex;
  dst->d_num_edge = src.num_edge;
#ifdef LAMBDA_TEST_SHORTEST_PATH
  //// TODO(laigd): add user defined members
  dst->d_source = src.source;
#else
$$G[[dst->d_<GP_NAME> = src.<GP_NAME>;]]
#endif
}

#ifdef LAMBDA_DEBUG
void GlobalManager::DebugOutput(const Global &global) {
  cout << LAMBDA_HEADER << "[Global]" << endl;
  cout << LAMBDA_HEADER
      << "num_vertex: " << global.d_num_vertex << ", "
      << "num_edge: " << global.d_num_edge
      //// TODO(laigd): add user defined members
#ifdef LAMBDA_TEST_SHORTEST_PATH
      << ", " << "source: " << global.d_source
#else
$$G[[<< ", " << "<GP_NAME>: " << global.d_<GP_NAME>]]
#endif
      << endl;
}
#endif

#ifdef LAMBDA_DEBUG
#undef LAMBDA_HEADER
#endif
