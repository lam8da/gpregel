#include "hip/hip_runtime.h"
#include <algorithm>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <fstream>
#include <iostream>
#include <map>
#include <set>
#include <string>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#include "sssp.h"

using std::cerr;
using std::cin;
using std::cout;
using std::endl;
using std::ifstream;
using std::map;
using std::ofstream;
using std::set;
using std::string;
using std::vector;

__global__ void SsspKernel1(
    const unsigned int numv,
    const unsigned int *va,
    const unsigned int *ea,
    const unsigned int *wa,
    bool *ma,
    unsigned int *ca,
    unsigned int *ua) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < numv) {
    if (ma[tid]) {
      ma[tid] = false;
      const unsigned int end = va[tid];
      for (int i = (tid == 0 ? 0 : va[tid - 1]); i < end; ++i) {
        unsigned int nid = ea[i];
        unsigned int w = ca[tid] + wa[i];
        atomicMin(ua + nid, w);
      }
    }
  }
}

__global__ void SsspKernel2(
    const unsigned int numv,
    const unsigned int *va,
    const unsigned int *ea,
    const unsigned int *wa,
    bool *ma,
    unsigned int *ca,
    unsigned int *ua) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < numv) {
    if (ca[tid] > ua[tid]) {
      ca[tid] = ua[tid];
      ma[tid] = true;
      ma[numv] = true;  // notify host that the iteration is continueing.
    }
    ua[tid] = ca[tid];
  }
}

unsigned int Sssp(
    const unsigned int numv,
    const unsigned int nume,
    const unsigned int source,
    const unsigned int *h_va,
    const unsigned int *h_ea,
    const unsigned int *h_wa,
    unsigned int *h_ca,
    float *duration) {
  unsigned int *d_va = NULL;
  unsigned int *d_ea = NULL;
  unsigned int *d_wa = NULL;
  bool *d_ma = NULL;
  unsigned int *d_ca = NULL;
  unsigned int *d_ua = NULL;

  const unsigned int vsize = numv * sizeof(unsigned int);
  const unsigned int esize = nume * sizeof(unsigned int);
  const unsigned int bsize = numv * sizeof(bool);

  checkCudaErrors(hipMalloc((void **)&d_va, vsize));
  checkCudaErrors(hipMalloc((void **)&d_ea, esize));
  checkCudaErrors(hipMalloc((void **)&d_wa, esize));
  checkCudaErrors(hipMalloc((void **)&d_ma, bsize + sizeof(bool)));
  checkCudaErrors(hipMalloc((void **)&d_ca, vsize));
  checkCudaErrors(hipMalloc((void **)&d_ua, vsize));

  checkCudaErrors(hipMemcpy(d_va, h_va, vsize, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_ea, h_ea, esize, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_wa, h_wa, esize, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemset(d_ma, 0, bsize + sizeof(bool)));
  checkCudaErrors(hipMemset(d_ca, 0xff, vsize));
  checkCudaErrors(hipMemset(d_ua, 0xff, vsize));

  bool boolean = true;
  checkCudaErrors(hipMemcpy(d_ma + source, &boolean, sizeof(bool), hipMemcpyHostToDevice));
  unsigned int source_dist = 0;
  checkCudaErrors(hipMemcpy(d_ca + source, &source_dist, sizeof(unsigned int), hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_ua + source, &source_dist, sizeof(unsigned int), hipMemcpyHostToDevice));

  const unsigned int threads_per_block = 128;
  const unsigned int blocks_per_grid =(numv + threads_per_block - 1) / threads_per_block;

  // ------------------------- start profiling -------------------------
  StopWatchInterface *timer;
  sdkCreateTimer(&timer);
  sdkResetTimer(&timer);
  sdkStartTimer(&timer);

  unsigned int superstep = 0;
  while (true) {
    boolean = false;
    checkCudaErrors(hipMemcpy(d_ma + numv, &boolean, sizeof(bool), hipMemcpyHostToDevice));

    SsspKernel1<<<blocks_per_grid, threads_per_block>>>(numv, d_va, d_ea, d_wa, d_ma, d_ca, d_ua);
    SsspKernel2<<<blocks_per_grid, threads_per_block>>>(numv, d_va, d_ea, d_wa, d_ma, d_ca, d_ua);
    ++superstep;

    checkCudaErrors(hipMemcpy(&boolean, d_ma + numv, sizeof(bool), hipMemcpyDeviceToHost));
    if (!boolean) break;
  }

  sdkStopTimer(&timer);
  *duration = sdkGetTimerValue(&timer);
  sdkDeleteTimer(&timer);
  // ------------------------- end profiling -------------------------

  checkCudaErrors(hipMemcpy(h_ca, d_ca, vsize, hipMemcpyDeviceToHost));

  checkCudaErrors(hipFree(d_va));
  checkCudaErrors(hipFree(d_ea));
  checkCudaErrors(hipFree(d_wa));
  checkCudaErrors(hipFree(d_ma));
  checkCudaErrors(hipFree(d_ca));
  checkCudaErrors(hipFree(d_ua));
  return superstep;
}

//argv[1]: gpu_id
int main(int argc, char **argv) {
  unsigned int gpuid = atoi(argv[1]);
  checkCudaErrors(hipSetDevice(gpuid));

  HostGraphGlobal global;
  vector<HostGraphVertex> vertex_vec;
  vector<HostGraphEdge> edge_vec;

  Read(global, vertex_vec, edge_vec);

  // ------------------------- start cpu -------------------------
  // not necessary, we assume that the vertexes are ordered in input file.
  // std::sort(vertex_vec.begin(), vertex_vec.end());
  std::sort(edge_vec.begin(), edge_vec.end());
  CpuAlgorithm(999999999, global, vertex_vec, edge_vec);

  // ------------------------- start gpu -------------------------
  unsigned int *h_va = new unsigned int[global.num_vertex];
  unsigned int *h_ea = new unsigned int[global.num_edge];
  unsigned int *h_wa = new unsigned int[global.num_edge];
  unsigned int *h_ca = new unsigned int[global.num_vertex];
  for (unsigned int i = 0; i < global.num_vertex; ++i) {
    h_va[i] = vertex_vec[i].sum_out_edge_count;
  }
  for (unsigned int i = 0; i < global.num_edge; ++i) {
    h_ea[i] = edge_vec[i].to;
    h_wa[i] = edge_vec[i].weight;
  }

  float duration = 0;
  unsigned int superstep = Sssp(global.num_vertex, global.num_edge, global.source, h_va, h_ea, h_wa, h_ca, &duration);

  // ------------------------- check result -------------------------
  bool correct = true;
  for (unsigned int i = 0; i < global.num_vertex; ++i) {
    if (h_ca[i] != vertex_vec[i].dist) {
      correct = false;

      ofstream out("gpuout-456767321776");
      out << "gpu result:" << endl;
      for (unsigned int j = 0; j <global.num_vertex; ++j) {
        out << h_ca[j] << endl;
      }
      out << endl;
      out.close();

      out.open("cpuout-456767321776");
      out << "cpu result:" << endl;
      for (unsigned int j = 0; j <global.num_vertex; ++j) {
        out << vertex_vec[j].dist << endl;
      }
      out << endl;
      out.close();
      break;
    }
  }

  delete[] h_va;
  delete[] h_ea;
  delete[] h_wa;
  delete[] h_ca;

  // ------------------------- output -------------------------
  cout << "check result: " << (correct ? "correct, " : "WRONG!!!!! ")
       << "source: " << global.source << ", "
       << "superstep: " << superstep << ", "
       << "gpu_duration: " << duration << " ms."
       << endl;

  checkCudaErrors(hipDeviceReset());
  return 0;
}

